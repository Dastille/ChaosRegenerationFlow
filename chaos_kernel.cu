
#include <hip/hip_runtime.h>
extern "C" __global__ void chaos_gen(unsigned long long seed, unsigned short tweak, size_t len, unsigned char *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        unsigned long long val = seed + idx;
        val = (val * 314159 + 299792458 + tweak) % 256;
        output[idx] = (unsigned char)val;
    }
}
